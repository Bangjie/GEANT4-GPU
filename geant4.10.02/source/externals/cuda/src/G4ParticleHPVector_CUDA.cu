#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"

/***********************************************
*   Device Methods
***********************************************/
__global__ void GetMinIndices_CUDA(G4ParticleHPDataPoint *d_theData, int nEntries, 
                                   double *d_energiesIn_xSecsOut, int numQueries, int *d_minIndices) {
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int stepSize = (int)sqrt((float)nEntries);

  if (idx < numQueries) {
    int i = 0;
    double e = d_energiesIn_xSecsOut[idx];
    
    for (i = 0; i < nEntries; i += stepSize) {
      if (d_theData[i].energy >= e) {
        break;
      }
    }
    
    i = (i - (stepSize - 1) >= 0) ? i - (stepSize - 1) : 0; 
    for (; i < nEntries; i++) {
      if (d_theData[i].energy >= e) {
        break;
      }
    }

    d_minIndices[idx] = i;
  }
}

void G4ParticleHPVector_CUDA::SetInterpolationManager(G4InterpolationManager & aManager) {
  theManager = aManager;
}
void G4ParticleHPVector_CUDA::SetInterpolationManager(const G4InterpolationManager & aManager) {
  theManager = aManager;
}

double getWallTime() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
}

/***********************************************
*   Host Methods
***********************************************/
void G4ParticleHPVector_CUDA::GetXsecList(G4double* energiesIn_xSecsOut, G4int numQueries, G4ParticleHPDataPoint* theData, G4int nEntries) {  
  if (nEntries == 0) {
    for (int i = 0; i < numQueries; i++) {
      energiesIn_xSecsOut[i] = 0.0;
    }
    return;
  }

  G4ParticleHPDataPoint * d_theData;
  G4double              * d_energiesIn_xSecsOut;
  G4int                 * d_minIndices;
  
  hipMalloc((void**)&d_theData,             sizeof(G4ParticleHPDataPoint)            * nEntries);
  hipMalloc((void**)&d_energiesIn_xSecsOut, sizeof(G4double)            * numQueries);
  hipMalloc((void**)&d_minIndices,          sizeof(G4int)               * numQueries);
  G4int *minIndices = (G4int*)malloc(numQueries * sizeof(G4int));

  hipMemcpy(d_theData, theData, sizeof(G4ParticleHPDataPoint) * nEntries, hipMemcpyHostToDevice);
  hipMemcpy(d_energiesIn_xSecsOut, energiesIn_xSecsOut, sizeof(G4double) * numQueries, hipMemcpyHostToDevice);
  
  // need to add 1 block if doesn't divide evenly (e.g 32 T_P_B, 36 numQueries we need 1+1=2 blocks to get those last 4 queries)
  int numBlocksSingleElement = numQueries/THREADS_PER_BLOCK + ((numQueries % THREADS_PER_BLOCK == 0) ? 0 : 1);

  GetMinIndices_CUDA <<<numBlocksSingleElement, THREADS_PER_BLOCK>>>
    (d_theData, nEntries, d_energiesIn_xSecsOut, numQueries, d_minIndices);

  hipMemcpy(minIndices, d_minIndices, sizeof(G4int) * numQueries, hipMemcpyDeviceToHost);

  for (int i = 0; i < numQueries; i++) {
    int minIndex = minIndices[i];
   
    G4int low = minIndex - 1;
    G4int high = minIndex;
    G4double e = energiesIn_xSecsOut[i];
    
    if (minIndex == 0)
    {
      low = 0;
      high = 1;
    }
    else if (minIndex == nEntries)
    {
      low = nEntries - 2;
      high = nEntries - 1;
    }

    if (e < theData[nEntries-1].GetX())
    {
      if (theData[high].GetX() != 0 
        &&(std::abs((theData[high].GetX() - theData[low].GetX()) / theData[high].GetX()) < 0.000001))
      {
        energiesIn_xSecsOut[i] = theData[low].GetY();
      }
      else
      {
        energiesIn_xSecsOut[i] = theInt.Interpolate(theManager.GetScheme(high), e, 
                               theData[low].GetX(), theData[high].GetX(),
                               theData[low].GetY(), theData[high].GetY());
      }
    }
    else
    {
      energiesIn_xSecsOut[i] = theData[nEntries-1].GetY();
    }
  }

  hipFree(d_theData);
  hipFree(d_energiesIn_xSecsOut);
  hipFree(d_minIndices);
  free(minIndices);
}