#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"

/***********************************************
*   CUDA functions
***********************************************/
__global__ void SetValueTo_CUDA(int *addressToSet, int value) {
    *(addressToSet) = value;
}

// http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
__device__ double AtomicAdd_CUDA(double* address, double val) { 
    unsigned long long int* address_as_ull = (unsigned long long int*)address; 
    unsigned long long int old = *address_as_ull, assumed; 
    do { 
        assumed = old; 
        old = atomicCAS(address_as_ull, assumed, 
            __double_as_longlong(val + __longlong_as_double(assumed))); 
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
    } while (assumed != old); 
    return __longlong_as_double(old); 
}

__global__ void CopyDataPointsToBuffer_CUDA(G4ParticleHPDataPoint * fromBuffer, G4ParticleHPDataPoint * toBuffer, G4int nEntries) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nEntries) {
        toBuffer[i].energy = fromBuffer[i].energy;
        toBuffer[i].xSec = fromBuffer[i].xSec;
    }
}

/***********************************************
*   Constructors, Deconstructors
***********************************************/
G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA()      { 
    nPoints = 20;
    hipMalloc(&d_theData, nPoints*sizeof(G4ParticleHPDataPoint));
    nEntries = 0;
    Verbose = 0;
    d_theIntegral = 0; // TODO: cuda malloc ?
    totalIntegral = -1;
    isFreed = 0;
    maxValue = -DBL_MAX;
    the15percentBorderCash = -DBL_MAX;
    the50percentBorderCash = -DBL_MAX;
    label = -DBL_MAX;
}

G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA(int n) { 
    nPoints = std::max(n,20);
    hipMalloc(&d_theData, nPoints*sizeof(G4ParticleHPDataPoint));
    nEntries = 0;
    Verbose = 0;
    d_theIntegral = 0; // TODO: cuda malloc ?
    totalIntegral = -1;
    isFreed = 0;
    maxValue = -DBL_MAX;
    the15percentBorderCash = -DBL_MAX;
    the50percentBorderCash = -DBL_MAX;
    label = -DBL_MAX;
}

G4ParticleHPVector_CUDA::~G4ParticleHPVector_CUDA() {
    if (d_theData) {
        hipFree(d_theData);
        d_theData = NULL;
    }
    if (d_theIntegral) {
       hipFree(d_theIntegral);
       d_theIntegral = NULL;
    }
    isFreed = 1;
}

/******************************************
* Getters from .hh that use CUDA
******************************************/
const G4ParticleHPDataPoint & G4ParticleHPVector_CUDA::GetPoint(G4int i) {
    G4ParticleHPDataPoint point;
    hipMemcpy(&point, &d_theData[i], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    return point;
}

G4double G4ParticleHPVector_CUDA::GetEnergy(G4int i) {
    G4double energy;
    hipMemcpy(&energy, &d_theData[i].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    return energy;
}

G4double G4ParticleHPVector_CUDA::GetX(G4int i) {
    G4double energy;
    hipMemcpy(&energy, &d_theData[i].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    return energy;
}

G4double G4ParticleHPVector_CUDA::GetXsec(G4int i) {
    G4double xSec;
    hipMemcpy(&xSec, &d_theData[i].xSec, sizeof(G4double), hipMemcpyDeviceToHost);
    return xSec;
}

// TODO: Port Me (requires 1st element predicate alg.)
G4double G4ParticleHPVector_CUDA::GetXsec(G4double e, G4int min) {
    return 0;
}

G4double G4ParticleHPVector_CUDA::GetY(G4double x) {
    return GetXsec(x);
}

G4double G4ParticleHPVector_CUDA::GetY(G4int i) {
    G4double xSec;
    hipMemcpy(&xSec, &d_theData[i].xSec, sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    return xSec;
}

// TODO: Port Me (requires interpolation)
G4double G4ParticleHPVector_CUDA::GetMeanX() {
    return 0;
}

/******************************************
* Setters from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::SetData(G4int i, G4double x, G4double y) {
    Check(i);
    G4ParticleHPDataPoint point;
    point.energy = x;
    point.xSec = y;
    hipMemcpy(&d_theData[i], &point, sizeof(G4ParticleHPDataPoint), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetX(G4int i, G4double e) {
    Check(i);
    hipMemcpy(&d_theData[i].energy, &e, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetEnergy(G4int i, G4double e) {
    Check(i);
    hipMemcpy(&d_theData[i].energy, &e, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetY(G4int i, G4double x) {
    Check(i);
    hipMemcpy(&d_theData[i].xSec, &x, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetXsec(G4int i, G4double x) {
    Check(i);
    hipMemcpy(&d_theData[i].xSec, &x, sizeof(G4double), hipMemcpyHostToDevice);
}


/******************************************
* Computations from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::Init(std::istream & aDataFile, G4double ux, G4double uy) {
    G4int total;
    aDataFile >> total;
    if (d_theData) {
        hipFree(d_theData);
    }
    hipMalloc(&d_theData, sizeof(G4ParticleHPDataPoint) * total);
    nPoints = total;
    nEntries = 0;
    theManager.Init(aDataFile);
    Init(aDataFile, total, ux, uy);
}

void G4ParticleHPVector_CUDA::CleanUp() {
    printf("\nCUDA - CleanUp (nEntries: %d", nEntries);
    nEntries = 0;
    theManager.CleanUp();
    maxValue = -DBL_MAX;
    if (d_theIntegral) {
        hipFree(d_theIntegral);
        d_theIntegral = NULL;
    }
}

__global__ void SampleLinFindLastIndex_CUDA(G4double * theIntegral, int rand, int * resultIndex, int nEntries) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nEntries) {
        return;
    }
    if (i > *resultIndex && theIntegral[i]/theIntegral[nEntries-1] < rand) {
        atomicMax(resultIndex, i);
    }
}
__global__ void SampleLinGetValues(G4ParticleHPDataPoint * theData, G4double * theIntegral, G4double * d_vals, G4int i) {
    // d_vals = [x1,x2,y1,y2]
    switch(threadIdx.x) {
        case 0: 
            d_vals[0] = theIntegral[i-1];
            break;
        case 1:
            d_vals[1] = theIntegral[i];
            break;
        case 2:
            d_vals[2] = theData[i-1].energy;
            break;
        case 3:
            d_vals[3] = theData[i].energy;
            break;
        default:
            printf("\nError -- invalid thread id in SampleLinGetValues(), returning");
    }
}

G4double G4ParticleHPVector_CUDA::SampleLin() {
    printf("\nCUDA - SampleLin (nEntries: %d", nEntries);
    G4double result;
    if (!d_theIntegral) {
        IntegrateAndNormalise();
    }

    if (GetVectorLength() == 1) {
        hipMemcpy(&result, &d_theData[0].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    }
    else {
        // TODO: change to G4UniformRand
        G4double randNum = (G4double)rand() / (G4double)RAND_MAX; 

        int *d_resultIndex;
        hipMalloc(&d_resultIndex, sizeof(int));
        SetValueTo_CUDA<<<1,1>>> (d_resultIndex, INT_MAX);

        int nBlocks = GetNumBlocks(nEntries);
        SampleLinFindLastIndex_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (d_theIntegral, randNum, d_resultIndex, nEntries);
        
        G4int i = 0;
        hipMemcpy(&i, d_resultIndex, sizeof(G4int), hipMemcpyDeviceToHost);
        if (i != GetVectorLength() - 1) {
            i++;
        }

        // vals = [x1, x2, y1, y2]
        G4double* d_vals;
        hipMalloc(&d_vals, 4*sizeof(G4double));
        SampleLinGetValues<<<1, 4>>>(d_theData, d_theIntegral, d_vals, i);
        
        G4double vals[4];
        hipMemcpy(vals, d_vals, 4*sizeof(G4double), hipMemcpyDeviceToHost);
        
        result = theLin.Lin(randNum, vals[0], vals[1], vals[2], vals[3]);
        
        hipFree(d_resultIndex);
        hipFree(d_vals);
        free(vals);
    }

    return result;
}

// TODO: Port Me (should return d_theIntegral, but how do we return somethign we don't have ref to?)
G4double * G4ParticleHPVector_CUDA::Debug() {
    printf("\nDEBUG NOT YET IMPLEMENTED");
    return 0;
}

// TODO: test that this gives same results
__global__ void Integrate_CUDA(G4ParticleHPDataPoint * theData, G4double * sum, G4InterpolationManager theManager) {
    G4int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i == 0) {
        return;
    }

    if (abs((theData[i].energy - theData[i-1].energy) / theData[i].energy) > 0.0000001) {
        G4double x1 = theData[i-1].energy;
        G4double x2 = theData[i].energy;
        G4double y1 = theData[i-1].xSec;
        G4double y2 = theData[i].xSec;

        double toAdd = 0;
        G4InterpolationScheme aScheme = theManager.GetScheme(i);
        if (aScheme == LINLIN || aScheme == CLINLIN || aScheme == ULINLIN) {
            toAdd += 0.5 * (y2+y1) * (x2-x1);
        }
        else if (aScheme == LINLOG || aScheme == CLINLOG || aScheme == ULINLOG) {
            G4double a = y1;
            // G4double b = (y2-y1) / (G4Log(x2) - G4Log(x1));
            // toAdd += (a-b) * (x2-x1) + b*(x2 * G4Log(x2) - x1 * G4Log(x1));
            
            // NOTE: cuda's log function requires compute capability >= 3.0 for double precision
            // make sure you are compiling for 3.0 (nvcc -arch sm_30)
            G4double b = (y2 - y1) / (log(x2) - log(x1));
            toAdd += (a-b) * (x2-x1) + b*(x2 * log(x2) - x1 * log(x1));
        }
        else if (aScheme == LOGLIN || aScheme == CLOGLIN || aScheme == ULOGLIN) {
            // G4double a = G4Log(y1);
            // G4double b = (G4Log(y2)-G4Log(y1))/(x2-x1);
            // toAdd += (G4Exp(a)/b) * (G4Exp(b * x2) - G4Exp(b * x1));

            // NOTE: cuda's log function requires compute capability >= 3.0 for double precision
            // make sure you are compiling for 3.0 (nvcc -arch sm_30)
            G4double a = log(y1);
            G4double b = (log(y2) - log(y1)) / (x2-x1);
            // toAdd += (G4Exp(a) / b) * (G4Exp(b * x2) - G4Exp(b * x1));

        }
        else if (aScheme == HISTO || aScheme == CHISTO || aScheme == UHISTO) {
            toAdd += y1 * (x2-x1);
        }
        else if (aScheme == LOGLOG || aScheme == CLOGLOG || aScheme == ULOGLOG) {
            // G4double a = G4Log(y1);
            // G4double b = (G4Log(y2) - G4Log(y1)) / (G4Log(x2) - G4Log(x1));
            // toAdd += (G4Exp(a)/(b+1)) * (G4Pow::GetInstance()->powA(x2,b+1) - G4Pow::GetInstance()->powA(x1,b+1));

            // NOTE: cuda's log function requires compute capability >= 3.0 for double precision
            // make sure you are compiling for 3.0 (nvcc -arch sm_30)
            G4double a = log(y1);
            G4double b = (log(y2) - log(y1)) / (log(x2) - log(x1));
            toAdd += (G4Exp(a)/(b+1)) * (pow(x2,b+1) - pow(x1,b+1));
        }

        if (toAdd != 0) {
            AtomicAdd_CUDA(sum, toAdd);
        }
    }
}
void G4ParticleHPVector_CUDA::Integrate() {
    printf("\nCUDA - Integrate (nEntries: %d", nEntries);
    if (nEntries == 1) {
        totalIntegral = 0;
        return;
    }
    
    G4double *d_sum;
    hipMalloc(&d_sum, sizeof(G4double));
    Integrate_CUDA<<<1, nEntries>>>(d_theData, d_sum, theManager);
    totalIntegral = *(d_sum);
    hipFree(d_sum);
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::IntegrateAndNormalise() {

}

__global__ void Times_CUDA(G4double factor, G4ParticleHPDataPoint* theData, G4double* theIntegral, G4int nEntriesArg) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nEntriesArg) {
        return;
    }
    theData[tid].xSec = theData[tid].xSec * factor;
    theIntegral[tid] = theIntegral[tid] * factor;
}
void G4ParticleHPVector_CUDA::Times(G4double factor) {
	printf("\nCUDA - Times (nEntries: %d", nEntries);
	int nBlocks = GetNumBlocks(nEntries);
    Times_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (factor, d_theData, d_theIntegral, nEntries);
}

/******************************************
* Functions from .cc
******************************************/
__global__ void GetXSecFirstIndex_CUDA(G4ParticleHPDataPoint * theData, G4double e, int * resultIndex, int nEntries) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < nEntries && idx < *(resultIndex) && theData[idx].energy > e) {
        atomicMin(resultIndex, idx);
    }
}
G4double G4ParticleHPVector_CUDA::GetXsec(G4double e) {
	int *d_resultIndex;
	hipMalloc(&d_resultIndex, sizeof(int));
	SetValueTo_CUDA<<<1,1>>> (d_resultIndex, INT_MAX);
	
    int nBlocks = GetNumBlocks(nEntries);
    GetXSecFirstIndex_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (d_theData, e, d_resultIndex, nEntries);
    
    G4int i = 0;
    hipMemcpy(&i, d_resultIndex, sizeof(G4int), hipMemcpyDeviceToHost);
    
    G4int low = i - 1;
    G4int high = i;
    if (i == 0) {
        low = 0;
        high = 1;
    }
    else if (i == nEntries) {
        low = nEntries - 2;
        high = nEntries - 1;
    }

    G4double y;
    G4ParticleHPDataPoint lastPoint;
    hipMemcpy(&lastPoint, &d_theData[nEntries-1], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    
    if (e < lastPoint.energy) {
        G4ParticleHPDataPoint theDataLow;
        G4ParticleHPDataPoint theDataHigh;
        hipMemcpy(&theDataLow, &d_theData[low], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
        hipMemcpy(&theDataHigh, &d_theData[high], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);

        if ((theDataHigh.energy - theDataLow.energy) / theDataHigh.energy < 0.000001) {
            y = theDataLow.xSec;
        }
        else {
            y = theInt.Interpolate(theManager.GetScheme(high), e, 
                    theDataLow.energy, theDataHigh.energy,
                    theDataLow.xSec, theDataHigh.xSec);
        }
    }
    else {
        y = lastPoint.xSec;
    }
	
    hipFree(d_resultIndex);
    return y;
}

void G4ParticleHPVector_CUDA::Dump() {
    printf("\nCUDA - Dump (nEntries: %d", nEntries);

    // never called, so just copy all of theData to cpu and print it out (slow, but works)
    G4ParticleHPDataPoint *localTheData = (G4ParticleHPDataPoint*)malloc(nEntries * sizeof(G4ParticleHPDataPoint));
    hipMemcpy(localTheData, d_theData, nEntries * sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    
    std::cout << nEntries << std::endl;
    for (G4int i = 0; i < nEntries; i++) {
        std::cout << localTheData[i].GetX() << " ";
        std::cout << localTheData[i].GetY() << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;

    free(localTheData);
}

// TODO: Make me parallel (works, but is serial so memcpy's too much)
void G4ParticleHPVector_CUDA::ThinOut(G4double precision) {
    printf("\nCUDA - ThinOut (nEntries: %d", nEntries);
    if (GetVectorLength() == 0) {
      return;
    }

    G4ParticleHPDataPoint *localTheData = (G4ParticleHPDataPoint*)malloc(nEntries*sizeof(G4ParticleHPDataPoint));
    hipMemcpy(localTheData, d_theData, nEntries*sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    G4ParticleHPDataPoint *localBuffer = (G4ParticleHPDataPoint*)malloc(nPoints*sizeof(G4ParticleHPDataPoint));
    
    G4double x, x1, x2, y, y1, y2;
    G4int count = 0;
    G4int current = 2;
    G4int start = 1;

    // first element always goes and is never tested.
    localBuffer[0] = localTheData[0];
    // copyDataPointFromBufferToBuffer_CUDA<<<1,1>>> (d_theData, localBuffer, nEntries);

    while(current < GetVectorLength()) {
        x1 = localBuffer[count].GetX();
        y1 = localBuffer[count].GetY();
        x2 = localTheData[current].GetX();
        y2 = localTheData[current].GetY();
        
        for(G4int j=start; j<current; j++) {
            x = localTheData[j].GetX();
        
            if (x1-x2 == 0) {
                y = (y2+y1)/2.0;
            }
            else {
                y = theInt.Lin(x, x1, x2, y1, y2);
            }
            if (std::abs(y - localTheData[j].GetY()) > precision * y) {
                localBuffer[++count] = localTheData[current-1]; // for this one, everything was fine
                start = current; // the next candidate
                break;
            }
        }
        current++;
    }

    // the last one also always goes, and is never tested.
    count++;
    localBuffer[count] = localTheData[GetVectorLength() - 1];
    nEntries = count + 1;

    hipFree(d_theData);
    hipMemcpy(d_theData, localBuffer, nEntries * sizeof(G4ParticleHPDataPoint), hipMemcpyHostToDevice);

    free(localTheData);
    free(localBuffer);
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::Merge(G4InterpolationScheme aScheme, G4double aValue, G4ParticleHPVector_CUDA * active, G4ParticleHPVector_CUDA * passive) {
    printf("MERGE NOT YET IMPLEMENTED\n\n");
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Sample() {
    printf("SAMPLE NOT YET IMPLEMENTED\n\n");
    return 0;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Get15percentBorder() {
    printf("Get 15 NOT YET IMPLEMENTED\n\n");
    return 0;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Get50percentBorder() {
    printf("Get 50 NOT YET IMPLEMENTED\n\n");
    return 0;
}

void G4ParticleHPVector_CUDA::Check(G4int i) {
    if (i > nEntries) {
        // throw G4HadronicException(__FILE__, __LINE__, "Skipped some index numbers in G4ParticleHPVector");
    }
    if (i == nPoints) {
        nPoints = static_cast<G4int>(1.2 * nPoints);
        G4ParticleHPDataPoint* d_newTheData;
        hipMalloc(&d_newTheData, nPoints*sizeof(G4ParticleHPDataPoint));

        int nBlocks = GetNumBlocks(nEntries);
        CopyDataPointsToBuffer_CUDA<<<nBlocks,THREADS_PER_BLOCK>>> (d_theData, d_newTheData, nEntries);
        
        hipFree(d_theData);
        d_theData = d_newTheData;
    }
    
    if (i == nEntries) {
        nEntries = i + 1;
    }
}

// Note: Geant4 doesn't ever assign private variable theBlocked,
// which means their IsBlocked function always returns false
G4bool G4ParticleHPVector_CUDA::IsBlocked(G4double aX) {
    return false;
}

// G4ParticleHPVector_CUDA:: & operatorPlus (G4ParticleHPVector & left, G4ParticleHPVector & right) { }
