#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.h"

/***********************************************
*   CUDA functions
***********************************************/
__global__ void cudaTimes(double factor, G4ParticleHPDataPoint* cudaTheData, double* cudaTheIntegral) {
    int tid = blockIdx.x;
    cudaTheData[tid].xSec = cudaTheData[tid].xSec*factor;
    cudaTheIntegral[tid] = cudaTheIntegral[tid]*factor;
}

/***********************************************
*   Constructors, Setters
***********************************************/
G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA()      { }
G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA(int n) { }

G4ParticleHPVector_CUDA::~G4ParticleHPVector_CUDA() {
    if (cudaTheData) {
        hipFree(cudaTheData);
    }
    if (cudaTheIntegral) {
       hipFree(cudaTheIntegral);
    }
}

void G4ParticleHPVector_CUDA::SetNEntries(int * nEntriesPointer) {
    nEntries = nEntriesPointer;
}

void G4ParticleHPVector_CUDA::SetNPoints(int * nPointsPointer) {
    nPoints = nPointsPointer;
}

void G4ParticleHPVector_CUDA::SetTheData(G4ParticleHPDataPoint ** theDataPointer) {
    theData = theDataPointer;
}

void G4ParticleHPVector_CUDA::SetTheIntegral(double ** theIntegralPointer) {
    theIntegral = theIntegralPointer;
}

void G4ParticleHPVector_CUDA::SetTheDataChangedOnCpu() {
    theDataChangedOnCpuBool = true;
    theDataChangedOnGpuBool = false;
}

void G4ParticleHPVector_CUDA::SetTheDataChangedOnGpu() {
    theDataChangedOnCpuBool = false;
    theDataChangedOnGpuBool = true;
}

void G4ParticleHPVector_CUDA::CopyTheDataToGpuIfChanged() {
    if (theDataChangedOnCpuBool && theDataChangedOnGpuBool) {
        printf("BIG ERROR in CopyTheDataToGpuIfChanged: theDataChangedOnCpu and theDataChangedOnGpu are both true!\n\n");
        return;
    }

    if (theDataChangedOnCpuBool) {
        int theDataSize = *(nEntries) * sizeof(G4ParticleHPDataPoint);  
        if (cudaTheDataSize != theDataSize) {
            if (cudaTheDataSize != 0) {
                hipFree(cudaTheData);
            }
            hipMalloc(&cudaTheData, theDataSize);
            cudaTheDataSize = theDataSize;
        }
        hipMemcpy(cudaTheData, *(theData), theDataSize, hipMemcpyHostToDevice);
        
        theDataChangedOnCpuBool = false;
        theDataChangedOnGpuBool = false;
    }
}

void G4ParticleHPVector_CUDA::CopyTheDataToCpuIfChanged() {
    if (theDataChangedOnCpuBool && theDataChangedOnGpuBool) {
        printf("BIG ERROR in CopyTheDataToCpuIfChanged: theDataChangedOnCpu and theDataChangedOnGpu are both true!\n\n");
        return;
    }

    if (theDataChangedOnGpuBool) {
        // gpu never changes size of theData, so just copy it over
        int theDataSize = *(nEntries) * sizeof(G4ParticleHPDataPoint);  
        hipMemcpy(*(theData), cudaTheData, theDataSize, hipMemcpyDeviceToHost);
        
        theDataChangedOnCpuBool = false;
        theDataChangedOnGpuBool = false;
    }
}

/***********************************************
*   Ported Functions
***********************************************/
void G4ParticleHPVector_CUDA::Times(double factor) {
    CopyTheDataToGpuIfChanged();

    int theIntegralSize = *(nEntries) * sizeof(double);
    hipMalloc(&cudaTheIntegral, theIntegralSize);    
    hipMemcpy(cudaTheIntegral, *(theIntegral), theIntegralSize, hipMemcpyHostToDevice);
    
    cudaTimes<<<*(nEntries),1>>>(factor, cudaTheData, cudaTheIntegral);
    SetTheDataChangedOnGpu();

    hipMemcpy(*(theIntegral), cudaTheIntegral, theIntegralSize, hipMemcpyDeviceToHost);
}

double G4ParticleHPVector_CUDA::GetXsec(double e) {
    printf("\nGetXsec");
    return -1;
}
