#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"

/***********************************************
*   Device Methods
***********************************************/
__global__ void SetArrayTo(int *array, int length, int setValue)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    array[idx] = setValue;
  }
}

__global__ void GetXSecFirstIndexArray_CUDA(G4ParticleHPDataPoint *d_theData, G4double *d_queryList, int *d_resArray, int numThreads, int querySize, int nEntries)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = 0; i < querySize; i++){
    G4double queryEnergy = d_queryList[i];
    
    for (int j = idx; j < nEntries; j += numThreads) {
      if (d_theData[j].energy >  queryEnergy) {
        atomicMin(&d_resArray[i], j);
      }
    }
  }
}

__global__ void GetYForXSecArray_CUDA(G4ParticleHPDataPoint *theData, int nEntries,  int *indexArray, GetXsecResultStruct * d_resArray, G4int querySize){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < querySize) {
    G4int low = indexArray[idx] -1;
    G4int high = indexArray[idx];
    
    if (indexArray[idx] == 0) {
      low = 0;
      high = 1;
    } else if (indexArray[idx] == nEntries) {
      d_resArray[idx].y = theData[nEntries-1].xSec;
    }
    
    if ((theData[high].energy != 0) && (abs((theData[high].energy - theData[low].energy) / theData[high].energy) < 0.000001)) {
      d_resArray[idx].y = theData[low].xSec;
    } else {
      d_resArray[idx].y = -1;
      d_resArray[idx].pointLow.energy = theData[low].energy;
      d_resArray[idx].pointLow.xSec = theData[low].xSec;
      d_resArray[idx].pointHigh.energy = theData[high].energy;
      d_resArray[idx].pointHigh.xSec = theData[high].xSec;
      d_resArray[idx].indexHigh = high;
    }
  }
}

void G4ParticleHPVector_CUDA::SetInterpolationManager(G4InterpolationManager & aManager) {
  theManager = aManager;
}
void G4ParticleHPVector_CUDA::SetInterpolationManager(const G4InterpolationManager & aManager) {
  theManager = aManager;
}
/***********************************************
*   Host Methods
***********************************************/
void G4ParticleHPVector_CUDA::GetXsecList(G4double* energiesIn_xSecsOut, G4int numQueries, G4ParticleHPDataPoint* theData, G4int nEntries) {  
  // printf("CUDA -- GetXsecList declaring...\n");
  G4ParticleHPDataPoint * d_theData;
  G4double              * d_energiesIn_xSecsOut;
  G4int                 * d_minIndices;
  GetXsecResultStruct   * d_resArray;
  GetXsecResultStruct   * h_resArray = (GetXsecResultStruct*)malloc(sizeof(GetXsecResultStruct) * numQueries);
  
  // printf("CUDA -- GetXsecList mallocing...\n");
  hipMalloc((void**)&d_theData,             sizeof(G4double)            * nEntries);
  hipMalloc((void**)&d_energiesIn_xSecsOut, sizeof(G4double)            * numQueries);
  hipMalloc((void**)&d_minIndices,          sizeof(G4int)               * numQueries);
  hipMalloc((void**)&d_resArray,            sizeof(GetXsecResultStruct) * numQueries);
  // hipHostMalloc(&h_resArray,                sizeof(GetXsecResultStruct) * numQueries);

  // printf("CUDA -- GetXsecList memcpying...\n");
  hipMemcpy(d_theData,             theData,             sizeof(G4ParticleHPDataPoint) * nEntries,   hipMemcpyHostToDevice);
  hipMemcpy(d_energiesIn_xSecsOut, energiesIn_xSecsOut, sizeof(G4double)              * numQueries, hipMemcpyHostToDevice);

  // need to add 1 block if doesn't divide evenly (e.g 32 T_P_B, 36 numQueries we need 1+1=2 blocks to get those last 4 queries)
  int numBlocksSingleElement = numQueries/THREADS_PER_BLOCK + ((numQueries % THREADS_PER_BLOCK == 0) ? 0 : 1);
  
  // each thread will work on multiple elements
  int elementsPerThread = 2;
  int totalNumThreads = nEntries / elementsPerThread;
  int numBlocksMultipleElements = totalNumThreads / THREADS_PER_BLOCK + ((totalNumThreads % THREADS_PER_BLOCK == 0) ? 0 : 1);
  
  // printf("CUDA -- GetXsecList SetArrayTo....\n");
  // initialize each index in array to last index of theData
  SetArrayTo <<<numBlocksSingleElement, THREADS_PER_BLOCK>>> 
    (d_minIndices, numQueries, nEntries - 1);

  // printf("CUDA -- GetXsecList GetXSecFirstIndexArray_CUDA...\n");
  // populate minIndices with the index of the first data point in theData with minimum energy
  GetXSecFirstIndexArray_CUDA <<<numBlocksMultipleElements, THREADS_PER_BLOCK>>>
    (d_theData, d_energiesIn_xSecsOut, d_minIndices, totalNumThreads, numQueries, nEntries);
  
  // printf("CUDA -- GetYForXSecArray_CUDA...\n");
  // fill resArray with struct containing either result if computed directly, or data points needed for interpolation
  GetYForXSecArray_CUDA <<<numBlocksSingleElement, THREADS_PER_BLOCK>>>
    (d_theData, nEntries, d_minIndices, d_resArray, numQueries);
  
  // printf("CUDA -- memcpying back to CPU...\n");
  hipMemcpy(h_resArray, d_resArray, sizeof(GetXsecResultStruct)*numQueries, hipMemcpyDeviceToHost);
  
  // printf("CUDA -- Interpolating...\n");
  // interpolate the values (if needed) on CPU (for now)
  for (int i = 0; i < numQueries; i++) {
    // printf("going through array, i = %d\n", i);
    GetXsecResultStruct res = h_resArray[i];
    // printf("set res to h_resArray[%d]\n", i);
    if (res.y != -1) {
      // printf("res.y is not -1\n");
      energiesIn_xSecsOut[i] = res.y;
    } else {
      // printf("actually interpolating, indexHigh = %d, theInt null: %d, theManager null: %d\n", res.indexHigh, (&theInt == NULL), (&theManager == NULL));
      G4double y = theInt.Interpolate(theManager.GetScheme(res.indexHigh), energiesIn_xSecsOut[i],
                                      res.pointLow.energy, res.pointHigh.energy,
                                      res.pointLow.xSec, res.pointHigh.xSec);
      // printf("done the interpolation, y = %f\n", y);
      // if (nEntries == 1) {
      //   energiesIn_xSecsOut[i] = 0.0;
      // }
      energiesIn_xSecsOut[i] = y;
    }
  }

  // printf("CUDA -- freeing...\n");
  hipFree(d_theData);
  hipFree(d_energiesIn_xSecsOut);
  hipFree(d_minIndices);
  hipFree(d_resArray);
  free(h_resArray);
}
