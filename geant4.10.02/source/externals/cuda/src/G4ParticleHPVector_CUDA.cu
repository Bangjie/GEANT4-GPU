#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"
#include <time.h>
#include <hiprand/hiprand_kernel.h>

/***********************************************
*   CUDA functions
***********************************************/
__global__ void SetValueTo_CUDA(int *addressToSet, int value) {
    *(addressToSet) = value;
}
__global__ void SetValueTo_CUDA(G4double *addressToSet, G4double value) {
    *(addressToSet) = value;
}
__global__ void SetValueTo_CUDA(G4ParticleHPDataPoint *addressToSet, G4double energy,
    G4double xSec) {
    addressToSet->energy = energy;
    addressToSet->xSec = xSec;
}

__global__ void GetXSecFirstIndex_CUDA(G4ParticleHPDataPoint * theData, G4double e,
        int * resultIndex, int numThreads, int nEntries) {
    int start = (blockDim.x * blockIdx.x + threadIdx.x);
    for (int i = start; i < nEntries; i += numThreads) {
        if (theData[i].energy >= e) {
            atomicMin(resultIndex, i);
            return;
        }
    }
}

__global__ void GetYForXSec_CUDA(G4ParticleHPDataPoint * theData, G4double e,
    G4int * singleIntResult, GetXsecResultStruct * resultsStruct, int nEntries) {
    G4int low = *(singleIntResult) - 1;
    G4int high = *(singleIntResult);
    if (*(singleIntResult) == 0) {
        low = 0;
        high = 1;
    } else if (*(singleIntResult) == nEntries) {
        low = nEntries - 2;
        high = nEntries - 1;
    }

    if (e < theData[nEntries - 1].energy) {
        if ((theData[high].energy != 0) && (abs((theData[high].energy - theData[low].energy) / theData[high].energy) < 0.000001)) {
            resultsStruct->y = theData[low].xSec;
        }
        else {
            resultsStruct->y = -1;
            resultsStruct->pointLow.energy = theData[low].energy;
            resultsStruct->pointLow.xSec = theData[low].xSec;
            resultsStruct->pointHigh.energy = theData[high].energy;
            resultsStruct->pointHigh.xSec = theData[high].xSec;
            resultsStruct->indexHigh = high;
        }
    }
    else {
        resultsStruct->y = theData[nEntries - 1].xSec;
    }
}

__device__ G4double rand_CUDA() {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);
    return hiprand_uniform_double(&state);
}

__global__ void SetAllNegativeXsecToZero_CUDA(G4ParticleHPDataPoint * theData, int nEntriesArg) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < nEntriesArg) {
        if (theData[tid].xSec < 0) {
            theData[tid].xSec = 0;
        }
    }
}

// http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
__device__ double AtomicAdd_CUDA(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void CopyDataPointsToBuffer_CUDA(G4ParticleHPDataPoint * fromBuffer,
    G4ParticleHPDataPoint * toBuffer, G4int nEntries) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nEntries) {
        toBuffer[i].energy = fromBuffer[i].energy;
        toBuffer[i].xSec = fromBuffer[i].xSec;
    }
}

__global__ void CopyTheIntegralToBuffer_CUDA(G4double * fromBuffer, G4double * toBuffer,
    G4int nEntries) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nEntries) {
        toBuffer[i] = fromBuffer[i];
    }
}
  
/***********************************************
*   Constructors, Deconstructors
***********************************************/
G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA()      {
    PerformInitialization(20);
}

G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA(G4int n) {
    PerformInitialization(std::max(n,20));
}

void G4ParticleHPVector_CUDA::PerformInitialization(G4int n) {
    nPoints = n;
    isDataDirtyHost = true;
    isIntegralDirtyHost = true;
    
    // theData
    h_theData = (G4ParticleHPDataPoint*)malloc(nPoints * sizeof(G4ParticleHPDataPoint));
    nPointsDataHost = nPoints;
    hipMalloc((void**)&d_theData, nPoints * sizeof(G4ParticleHPDataPoint));
    
    // memory for function results on cuda and host
    hipMalloc((void**)&d_singleIntResult, sizeof(G4int));
    hipHostMalloc(&h_singleIntResult, sizeof(G4int));
    hipMalloc((void**)&d_singleDoubleResult, sizeof(G4double));
    hipHostMalloc(&h_singleDoubleResult, sizeof(G4double));
    hipMalloc((void**)&d_res, sizeof(GetXsecResultStruct));
    hipHostMalloc(&h_res, sizeof(GetXsecResultStruct));
    
    // theIntegral remains null until we call Integrate
    d_theIntegral = NULL;
    h_theIntegral = NULL;
    nPointsIntegralHost = 0;
    
    nEntries = 0;
    Verbose = 0;
    totalIntegral = -1;
    isFreed = 0;
    maxValue = -DBL_MAX;
    the15percentBorderCash = -DBL_MAX;
    the50percentBorderCash = -DBL_MAX;
    label = -DBL_MAX;
}

G4ParticleHPVector_CUDA::~G4ParticleHPVector_CUDA() {
    if (d_theData) {
        hipFree(d_theData);
        d_theData = NULL;
    }
    if (h_theData) {
        free(h_theData);
        h_theData = NULL;
    }
    if (d_theIntegral) {
       hipFree(d_theIntegral);
       d_theIntegral = NULL;
    }
    if (h_theIntegral) {
        free(h_theIntegral);
        h_theIntegral = NULL;
    }
    if (d_singleIntResult) {
        hipFree(d_singleIntResult);
        d_singleIntResult = NULL;
    }
    if (h_singleIntResult) {
        hipHostFree(h_singleIntResult);
        h_singleIntResult = NULL;
    }
    if (d_singleDoubleResult) {
        hipFree(d_singleDoubleResult);
        d_singleDoubleResult = NULL;
    }
    if (h_singleDoubleResult) {
        hipHostFree(h_singleDoubleResult);
        h_singleDoubleResult = NULL;
    }
    if (d_res) {
        hipFree(d_res);
        d_res = NULL;
    }
    if (h_res) {
        hipHostFree(h_res);
        h_res = NULL;
    }
    
    isFreed = 1;
}

void G4ParticleHPVector_CUDA::OperatorEquals(G4ParticleHPVector_CUDA * right) {
    totalIntegral = right->totalIntegral;
    nEntries = right->nEntries;
    nPoints = right->nPoints;

    int numBlocks = GetNumBlocks(nEntries);

    if (right->d_theIntegral != 0) {
        if (d_theIntegral) {
            free(d_theIntegral);
        }
        hipMalloc((void**)&d_theIntegral, nEntries * sizeof(G4double));
        CopyTheIntegralToBuffer_CUDA<<<numBlocks, THREADS_PER_BLOCK>>> (right->d_theIntegral, d_theIntegral, nEntries);
    }

    hipMalloc((void**)&d_theData, nPoints * sizeof(G4ParticleHPDataPoint));
    CopyDataPointsToBuffer_CUDA<<<numBlocks, THREADS_PER_BLOCK>>> (right->d_theData, d_theData, nEntries);

    theManager = right->theManager;
    label = right->label;

    Verbose = right->Verbose;
    the15percentBorderCash = right->the15percentBorderCash;
    the50percentBorderCash = right->the50percentBorderCash;

    isDataDirtyHost = true;
    isIntegralDirtyHost = true;
}

void G4ParticleHPVector_CUDA::CopyToCpuIfDirty() {
    if (isDataDirtyHost) {
        if (nPointsDataHost != nPoints) {
            h_theData = (G4ParticleHPDataPoint*)realloc(h_theData, nPoints * sizeof(G4ParticleHPDataPoint));
            if (!h_theData) { printf("\nMALLOC FAILED IN COPY DATA TO CPU"); }
            nPointsDataHost = nPoints;
        }
        hipError_t err = hipMemcpy(h_theData, d_theData, nEntries * sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
        if (err != 0) {
            printf("Error Found (theData): %s\n", hipGetErrorString(err));
        }
        isDataDirtyHost = false;
    }

    if (isIntegralDirtyHost) {
        // device theIntegral is null, make ours match
        if (d_theIntegral == NULL) {
            if (h_theIntegral) {
                free(h_theIntegral);
            }
            h_theIntegral = NULL;
            nPointsIntegralHost = 0;
        }
        
        // copy theIntegral from device
        else {
            if (nPointsIntegralHost != nPoints) {
                h_theIntegral = (G4double*)realloc(h_theIntegral, nPoints * sizeof(G4double));
                if (!h_theIntegral) { printf("\nREALLOC FAILED IN COPY INTEGRAL TO CPU"); }
                nPointsIntegralHost = nPoints;
            }
            hipError_t err = hipMemcpy(h_theIntegral, d_theIntegral, nEntries * sizeof(G4double), hipMemcpyDeviceToHost);
            if (err != 0) { printf("Error Found (integral): %s\n", hipGetErrorString(err)); }
        }
        isIntegralDirtyHost = false;
    }
}


/******************************************
* Getters from .hh that use CUDA
******************************************/
G4ParticleHPDataPoint & G4ParticleHPVector_CUDA::GetPoint(G4int i) {
    if (i < 0) {
        i = 0;
    } else if (i >= GetVectorLength()) {
        i = GetVectorLength() - 1;
    }
    G4ParticleHPDataPoint point;
    hipMemcpy(&point, &d_theData[i], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    G4ParticleHPDataPoint *res  = new G4ParticleHPDataPoint(point.energy, point.xSec);
    return *res;
}

G4double G4ParticleHPVector_CUDA::GetX(G4int i) {
    if (i < 0) {
        i = 0;
    }
    if (i >= GetVectorLength()) {
        i = GetVectorLength() - 1;
    }

    if (!isDataDirtyHost) {
        return h_theData[i].GetX();
    }
    hipMemcpy(h_singleDoubleResult, &d_theData[i].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    
    if (*(h_singleDoubleResult) != *(h_singleDoubleResult)) { printf("\nGetEnergy(%d) = %f, nEntries=%d", i, *h_singleDoubleResult, nEntries); }
    return *(h_singleDoubleResult);
}

G4double G4ParticleHPVector_CUDA::GetY(G4int i) {
    if (i < 0) {
        i = 0;
    }
    if (i >= GetVectorLength()) {
        i = GetVectorLength() - 1;
    }
    
    if (!isDataDirtyHost) {
        return h_theData[i].GetY();
    }
    hipMemcpy(h_singleDoubleResult, &d_theData[i].xSec, sizeof(G4double), hipMemcpyDeviceToHost);
    return *(h_singleDoubleResult);
}

G4double G4ParticleHPVector_CUDA::GetY(G4double x) {
    return GetXsec(x);
}

G4double G4ParticleHPVector_CUDA::GetXsec(G4double e, G4int min) {
    if (nEntries == 0) {
        return 0.0;
    }

    // Note: this was causing some crashing / finishing in 0.01s pre-Mar-3 commit, if it crops up
    // again try copying d_theData to a new local array and using that (every GetXSec call)
    // CopyToCpuIfDirty();
    // if (nEntries == 1 || min >= nEntries) {
    //     return h_theData[0].GetY();
    // }

    // G4int i;
    // min = (min >= 0) ? min : 0;
    // for (i = min; i < nEntries; i++) {
    //     if (h_theData[i].GetX() >= e) {
    //         break;
    //     }
    // }

    // G4int low = i - 1;
    // G4int high = i;
    // if (i == 0) {
    //     low = 0;
    //     high = 1;
    // }
    // else if (i == nEntries) {
    //     low = nEntries - 2;
    //     high = nEntries - 1;
    // }

    // G4double y;
    // if (e < h_theData[nEntries-1].GetX()) {
    //     if (h_theData[high].GetX() != 0
    //             && (std::abs((h_theData[high].GetX() - h_theData[low].GetX()) / h_theData[high].GetX()) < 0.000001)) {
    //         y = h_theData[low].GetY();
    //     }
    //     else {
    //         y = theInt.Interpolate(theManager.GetScheme(high), e,
    //                                h_theData[low].GetX(), h_theData[high].GetX(),
    //                                h_theData[low].GetY(), h_theData[high].GetY());
    //     }
    // }
    // else {
    //     y = h_theData[nEntries-1].GetY();
    // }

    // return y;

     // ===== Run GetXSec using CUDA ===========================================
    SetValueTo_CUDA<<<1,1>>> (d_singleIntResult, nEntries);

    // GetXSecFirstIndex = 0.000005s
    int elementsPerThread = 2;
    int nBlocks = GetNumBlocks(nEntries / elementsPerThread);
    int numThreads = nBlocks * THREADS_PER_BLOCK;
    GetXSecFirstIndex_CUDA<<<nBlocks, THREADS_PER_BLOCK>>>
        (d_theData, e, d_singleIntResult, numThreads, nEntries);

    // GetYForXSec = 0.000005s
    GetYForXSec_CUDA<<<1,1>>> (d_theData, e, d_singleIntResult, d_res, nEntries);

    // Performing memcpy (singleIntResult) = 0.00003s
    // Performing memcpy (h_res) = 0.00004s
    hipMemcpy(h_res, d_res, sizeof(GetXsecResultStruct), hipMemcpyDeviceToHost);

    GetXsecResultStruct res = *(h_res);
    if (res.y != -1) {
        return res.y;
    }
    else {
        G4double y = theInt.Interpolate(theManager.GetScheme(res.indexHigh), e,
                res.pointLow.energy, res.pointHigh.energy,
                res.pointLow.xSec, res.pointHigh.xSec);
        if (nEntries == 1) {
            return 0.0;
        }
        return y;
    } 
    // ===================================================================== 
}

// TODO: Port Me (requires interpolation)
G4double G4ParticleHPVector_CUDA::GetMeanX() {
    printf("\nGETMEANX NOT YET IMPLEMENTED");
    return 0;
}


/******************************************
* Setters from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::SetData(G4int i, G4double x, G4double y) {
    if (i < 0) {
        return;
    }
    Check(i);
    SetValueTo_CUDA<<<1,1>>> (&d_theData[i], x, y);
    if (x != x || y != y) { printf("\nSetData got passed NAN!, SetData(%d, %0.5e, %0.5e)", i, x, y); }
    isDataDirtyHost = true;
}

void G4ParticleHPVector_CUDA::SetX(G4int i, G4double e) {
    if (i < 0) {
        return;
    }
    Check(i);
    SetValueTo_CUDA<<<1,1>>> (&d_theData[i].energy, e);
    if (e != e) { printf("\nSetX(%d) got passed NAN!!!", i); }
    isDataDirtyHost = true;
}

void G4ParticleHPVector_CUDA::SetY(G4int i, G4double x) {
    if (i < 0) {
        return;
    }
    Check(i);
    SetValueTo_CUDA<<<1,1>>> (&d_theData[i].xSec, x);
    if (x != x) { printf("\nSety(%d) got passed NAN!!!", i); }
    isDataDirtyHost = true;
}

void G4ParticleHPVector_CUDA::SetEnergy(G4int i, G4double e) {
    SetX(i,e);
}

void G4ParticleHPVector_CUDA::SetXsec(G4int i, G4double x) {
    SetY(i,x);
}


/******************************************
* Computations from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::Init(std::istream & aDataFile, G4int total, G4double ux, G4double uy) {
    G4double x, y;
    
    // TODO: change to realloc, had some problems when it was realloc before
    //h_theData = (G4ParticleHPDataPoint*)realloc(h_theData, total * sizeof(G4ParticleHPDataPoint));
    if (h_theData) {
        free(h_theData);
    }
    h_theData = (G4ParticleHPDataPoint*)malloc(total * sizeof(G4ParticleHPDataPoint));
    if (!h_theData) { printf("MALLOC FAILURE - 296"); }
    
    for (G4int i = 0; i < total; i++) {
        aDataFile >> x >> y;
        x *= ux;
        y *= uy;
        h_theData[i] = G4ParticleHPDataPoint(x,y);
    }
    nPoints = total;
    nEntries = total;
    nPointsDataHost = total;

    if (d_theData) {
        hipFree(d_theData);
    }
    hipMalloc((void**)&d_theData, nPoints * sizeof(G4ParticleHPDataPoint));
    hipMemcpy(d_theData, h_theData, nEntries * sizeof(G4ParticleHPDataPoint), hipMemcpyHostToDevice);

    isDataDirtyHost = false;
}

void G4ParticleHPVector_CUDA::Init(std::istream & aDataFile, G4double ux, G4double uy) {
    G4int total;
    aDataFile >> total;

    nPoints = total;
    nEntries = 0;
    theManager.Init(aDataFile);
    Init(aDataFile, total, ux, uy);
}

void G4ParticleHPVector_CUDA::CleanUp() {
    printf("\nCUDA - CleanUp (nEntries: %d", nEntries);
    nEntries = 0;
    theManager.CleanUp();
    maxValue = -DBL_MAX;
    if (d_theIntegral) {
        hipFree(d_theIntegral);
        d_theIntegral = NULL;
    }
    if (h_theIntegral) {
        free(h_theIntegral);
        h_theIntegral = NULL;
    }
    nPointsIntegralHost = 0;
    isDataDirtyHost = true;
    isIntegralDirtyHost = true;
}

__global__ void SampleLinFindLastIndex_CUDA(G4double * theIntegral, int rand, int * resultIndex,
    int nEntries) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nEntries) {
        return;
    }
    if (i > *(resultIndex) && (theIntegral[i] / theIntegral[nEntries-1]) < rand) {
        atomicMax(resultIndex, i);
    }
}

__global__ void SampleLinGetValues(G4ParticleHPDataPoint * theData, G4double * theIntegral,
    G4double * d_vals, G4int i) {
    // d_vals = [x1,x2,y1,y2]
    switch(threadIdx.x) {
        case 0:
            d_vals[0] = theIntegral[i-1];
            break;
        case 1:
            d_vals[1] = theIntegral[i];
            break;
        case 2:
            d_vals[2] = theData[i-1].energy;
            break;
        case 3:
            d_vals[3] = theData[i].energy;
            break;
        default:
            printf("\nError -- invalid thread id in SampleLinGetValues(), returning");
    }
}

G4double G4ParticleHPVector_CUDA::SampleLin() {
    // G4double result;
    // if (!d_theIntegral) {
    //     IntegrateAndNormalise();
    // }

    // if (GetVectorLength() == 1) {
    //     hipMemcpy(&result, &d_theData[0].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    // }
    // else {
    //     G4double randNum = GetUniformRand();

    //     int *d_resultIndex;
    //     hipMalloc((void**)&d_resultIndex, sizeof(int));
    //     SetValueTo_CUDA<<<1,1>>> (d_resultIndex, INT_MAX);

    //     int nBlocks = GetNumBlocks(nEntries);
    //     SampleLinFindLastIndex_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (d_theIntegral, randNum, d_resultIndex, nEntries);

    //     G4int i = 0;
    //     hipMemcpy(&i, d_resultIndex, sizeof(G4int), hipMemcpyDeviceToHost);
    //     if (i != GetVectorLength() - 1) {
    //         i++;
    //     }

    //     // vals = [x1, x2, y1, y2]
    //     G4double* d_vals;
    //     hipMalloc((void**)&d_vals, 4*sizeof(G4double));
    //     SampleLinGetValues<<<1, 4>>>(d_theData, d_theIntegral, d_vals, i);

    //     G4double vals[4];
    //     hipMemcpy(vals, d_vals, 4*sizeof(G4double), hipMemcpyDeviceToHost);

    //     result = theLin.Lin(randNum, vals[0], vals[1], vals[2], vals[3]);

    //     hipFree(d_resultIndex);
    //     hipFree(d_vals);
    // }
    // isIntegralDirtyHost = true;
    // return result;
    CopyToCpuIfDirty();
    
    G4double result;
    if (d_theIntegral == NULL) {
        IntegrateAndNormalise();
        CopyToCpuIfDirty(); // theIntegral changed
    }
    if (GetVectorLength() == 0) {
        result = 0;
    } else if (GetVectorLength()==1) {
        result = h_theData[0].GetX();
    } else {
        G4int i;
        G4double rand = GetUniformRand();

        for (i = GetVectorLength()-1; i >= 0 ; i--) {
            if(rand > h_theIntegral[i]/h_theIntegral[GetVectorLength()-1])
            break;
        }
        if (i!=GetVectorLength()-1) {
          i++;
        }

        G4double x1, x2, y1, y2;
        y1 = h_theData[i-1].GetX();
        x1 = h_theIntegral[i-1];
        y2 = h_theData[i].GetX();
        x2 = h_theIntegral[i];
        // not really necessary, since the case is excluded by construction
        if(std::abs((y2-y1)/y2)<0.0000001) {
            y1 = h_theData[i-2].GetX();
            x1 = h_theIntegral[i-2];
        }
        result = theLin.Lin(rand, x1, x2, y1, y2);
    }
    return result;
}

G4double * G4ParticleHPVector_CUDA::Debug() {
    if (!d_theIntegral) {
        return NULL;
    } else {
        CopyToCpuIfDirty();
        return h_theIntegral;
    }
}

// TODO: test that this gives same results
__global__ void Integrate_CUDA(G4ParticleHPDataPoint * theData, G4double * sum,
    G4InterpolationManager theManager) {
    G4int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i == 0) {
        return;
    }

    if (abs((theData[i].energy - theData[i-1].energy) / theData[i].energy) > 0.0000001) {
        G4double x1 = theData[i-1].energy;
        G4double x2 = theData[i].energy;
        G4double y1 = theData[i-1].xSec;
        G4double y2 = theData[i].xSec;

        double toAdd = 0;
        G4InterpolationScheme aScheme = theManager.GetScheme(i);

        // NOTE: cuda's log function requires compute capability >= 3.0 for double precision
        // make sure you are compiling for 3.0 (nvcc -arch sm_30)
        if (aScheme == LINLIN || aScheme == CLINLIN || aScheme == ULINLIN) {
            toAdd += 0.5 * (y2+y1) * (x2-x1);
        }
        else if (aScheme == LINLOG || aScheme == CLINLOG || aScheme == ULINLOG) {
            G4double a = y1;
            G4double b = (y2 - y1) / (log(x2) - log(x1));
            toAdd += (a-b) * (x2-x1) + b*(x2 * log(x2) - x1 * log(x1));
        }
        else if (aScheme == LOGLIN || aScheme == CLOGLIN || aScheme == ULOGLIN) {
            G4double a = log(y1);
            G4double b = (log(y2) - log(y1)) / (x2-x1);
            toAdd += (G4Exp(a) / b) * (G4Exp(b * x2) - G4Exp(b * x1));
        }
        else if (aScheme == HISTO || aScheme == CHISTO || aScheme == UHISTO) {
            toAdd += y1 * (x2-x1);
        }
        else if (aScheme == LOGLOG || aScheme == CLOGLOG || aScheme == ULOGLOG) {
            G4double a = log(y1);
            G4double b = (log(y2) - log(y1)) / (log(x2) - log(x1));
            toAdd += (G4Exp(a)/(b+1)) * (pow(x2,b+1) - pow(x1,b+1));
        }

        if (toAdd != 0) {
            AtomicAdd_CUDA(sum, toAdd);
        }
    }
}

void G4ParticleHPVector_CUDA::Integrate() {
    printf("\nCUDA - Integrate (nEntries: %d)", nEntries);
    if (nEntries < 1) {
        totalIntegral = 0;
        return;
    }

    G4double *d_sum;
    hipMalloc((void**)&d_sum, sizeof(G4double));
    Integrate_CUDA<<<1, nEntries>>>(d_theData, d_sum, theManager);
    totalIntegral = *(d_sum);
    hipFree(d_sum);
}

__global__ void TimesTheIntegral_CUDA(G4double * theIntegral, G4int nEntries, G4double factor) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < nEntries) {
        theIntegral[tid] *= factor;
    }
}

void G4ParticleHPVector_CUDA::IntegrateAndNormalise() {
    printf("\nIntegrateAndNormalise (nEntries = %d)\n", nEntries) ;
    if (d_theIntegral != NULL) {
        return;
    }
    
    // allocate our arrays
    hipMalloc((void**)&d_theIntegral, nEntries * sizeof(G4double));
    h_theIntegral = (double*)malloc(nEntries * sizeof(G4double));
    nPointsIntegralHost = nEntries;

    if (nEntries == 1) {
        SetValueTo_CUDA<<<1,1>>> (&d_theIntegral[0], 1.0);
        isIntegralDirtyHost = true;
        return;
    }

    // copy theData locally, but theIntegral won't be copied
    isIntegralDirtyHost = false;
    CopyToCpuIfDirty();

    G4double sum = 0;
    G4double x1 = 0;
    G4double x0 = 0;
    h_theIntegral[0] = 0;
    
    for (G4int i = 1; i < GetVectorLength(); i++) {
        x1 = h_theData[i].GetX();
        x0 = h_theData[i-1].GetX();
        if (std::abs(x1 - x0) > std::abs(x1 * 0.0000001)) {
            G4InterpolationScheme aScheme = theManager.GetScheme(i);
            G4double y0 = h_theData[i-1].GetY();
            G4double y1 = h_theData[i].GetY();
            G4double integ = theInt.GetBinIntegral(aScheme,x0,x1,y0,y1);
            #if defined WIN32-VC
                if(!_finite(integ)){integ=0;}
            #elif defined __IBMCPP__
                if(isinf(integ)||isnan(integ)){integ=0;}
            #else
                if(std::isinf(integ)||std::isnan(integ)){integ=0;}
            #endif

            sum += integ;
        }
        h_theIntegral[i] = sum;
    }
    
    // copy our changes back to GPU
    hipMemcpy(d_theIntegral, h_theIntegral, nEntries * sizeof(G4double), hipMemcpyHostToDevice);

    G4double total = h_theIntegral[GetVectorLength()-1];
    int nBlocks = GetNumBlocks(nEntries);
    TimesTheIntegral_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (d_theIntegral, nEntries, 1.0/total);
    
    // will need to recopy after multiplying by factor
    isIntegralDirtyHost = true;
}

__global__ void Times_CUDA(G4double factor, G4ParticleHPDataPoint* theData, G4double* theIntegral,
                           G4int nEntriesArg) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // if (tid >= nEntriesArg) {
    //     if (theIntegral != 0 && tid == nEntriesArg) {
    //         // theIntegral[tid] = theIntegral[tid] * factor;
    //     }
    //     return;
    // }

    if (tid < nEntriesArg) {
        theData[tid].xSec = theData[tid].xSec * factor;
    }
}

void G4ParticleHPVector_CUDA::Times(G4double factor) {
    int nBlocks = GetNumBlocks(nEntries);
    Times_CUDA<<<nBlocks, THREADS_PER_BLOCK>>> (factor, d_theData, d_theIntegral, nEntries);

    isDataDirtyHost = true;
    isIntegralDirtyHost = true;
}

/******************************************
* Functions from .cc
******************************************/


G4double G4ParticleHPVector_CUDA::GetXsec(G4double e) {
    return GetXsec(e, 0);
}

//function used to set all elements in an array to the same value
__global__ void SetArrayTo(int *array, int arraySize, int setValue)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;	// determine threads ID
	if(idx < arraySize){
		array[idx] = setValue;
	}
	
}

__global__ void GetXSecFirstIndexArray_CUDA(G4ParticleHPDataPoint *theData_d, G4double *queryArray_d, int *resArray_d, int numThreads, int querySize, int nEntries)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;	// determine threads ID
	for (int i = 0; i < querySize; i++){// foreach query in the query List
		G4double queryEnergy = queryArray_d[i];
		for(int j = idx; j <= nEntries; j+= numThreads){// check threads designated chunk of data
			if(theData_d[j].energy >  queryEnergy){
				atomicMin(&resArray_d[i], j);
			}
		}
	}
}

__global__ void GetYForXSecArray_CUDA(G4ParticleHPDataPoint *theData, int nEntries,  int *indexArray, GetXsecResultStruct * d_resArray, G4int querySize){
	int idx = blockDim.x*blockIdx.x + threadIdx.x;	// determine threads ID
	if(idx < querySize){
		printf("indexArray[idx]: %i\n", indexArray[idx]);
		//queryArray_d[idx] = theData_d[resArray_d[idx]].xSec;
		G4int low = indexArray[idx] -1;
		G4int high = indexArray[idx];
		if(indexArray[idx] == 0){
			low = 0;
			high = 1;
		} else if(indexArray[idx] == nEntries){
			d_resArray[idx].y = theData[nEntries-1].xSec;
		}
		
		if ((theData[high].energy != 0) && (abs((theData[high].energy - theData[low].energy) / theData[high].energy) < 0.000001)) {
			d_resArray[idx].y = theData[low].xSec;
		}
		else {
			d_resArray[idx].y = -1;
			d_resArray[idx].pointLow.energy = theData[low].energy;
			d_resArray[idx].pointLow.xSec = theData[low].xSec;
			d_resArray[idx].pointHigh.energy = theData[high].energy;
			d_resArray[idx].pointHigh.xSec = theData[high].xSec;
			d_resArray[idx].indexHigh = high;
		}
	}
		
}
void G4ParticleHPVector_CUDA::GetXsecBuffer(G4double * queryList, G4int length){	
	//printf("function enter\n");
	GetXsecResultStruct * h_resArray;	// Array of result for host
	GetXsecResultStruct * d_resArray;	// Array for where the results are placed on the GPU
	G4double * d_queryList;				// device copy of the queryList
	G4int * d_indexArray;				// device array to keep track of indexs for xSec
	
	// Allocate memory for everything
	//printf("malloc\n");
	hipHostMalloc(&h_resArray, sizeof(GetXsecResultStruct) * length);
	hipMalloc((void**)&d_resArray, sizeof(GetXsecResultStruct) * length);
	hipMalloc((void**)&d_queryList, sizeof(G4double) * length);
	hipMalloc((void**)&d_indexArray, sizeof(G4int) * length);
	
	// Copy the queryList to the device
	//printf("memcpy\n");
	hipMemcpy(d_queryList, queryList, sizeof(G4double)*length, hipMemcpyHostToDevice);
	
	// Determine how many blocks we need to allocate 
	int block_size =  32;
	int arrayBlocks = length/block_size + (length%block_size == 0 ? 0:1); 		// For when we want 1 thread per index
	int dataChunk = 2;															// How many indexes a thread will check in theData
	int numThreads = nEntries/dataChunk;										// How many threads we will need for this
	int queryBlocks = dataChunk/block_size + (dataChunk%block_size == 0 ? 0:1);	// For when we want 1 thread for multiple indexes
	
	// Have GPU prime indexArray 
	//printf("Setting Array\n");
	SetArrayTo <<< arrayBlocks, block_size >>> (d_indexArray, length, nEntries -1);
	
	// Have GPU do its thing
	//printf("Starting Work\n");
	GetXSecFirstIndexArray_CUDA <<<queryBlocks, block_size >>> (d_theData, d_queryList, d_indexArray, numThreads, length, nEntries);
	
	// Use the result indexes to gather the data required for get the xSec
	//printf("Getting Results\n");
	GetYForXSecArray_CUDA <<< arrayBlocks, block_size >>>(d_theData, nEntries,  d_indexArray, d_resArray, length);
	
	// Copy the computed results back to the Host
	hipMemcpy(h_resArray, d_resArray, sizeof(GetXsecResultStruct)*length, hipMemcpyDeviceToHost);
	
	// need to interpolate the xSecs using CPU, for now
	//printf("interpolating xSecs\n");
	for(int i = 0; i < length; i++){
	    GetXsecResultStruct res = h_resArray[i];
		if (res.y != -1) {
			queryList[i] = res.y;
		}
		else {
			G4double y = theInt.Interpolate(theManager.GetScheme(res.indexHigh), queryList[i],
            res.pointLow.energy, res.pointHigh.energy,
            res.pointLow.xSec, res.pointHigh.xSec);
			if (nEntries == 1) {
				queryList[i] = 0.0;
			}
			queryList[i] = y;
		}
	}
	// Free the temporary data to avoid memory leaks
	 hipFree(d_resArray);
	 hipFree(d_queryList);
	 hipHostFree(h_resArray);
	 //printf("function done\n");
}


void G4ParticleHPVector_CUDA::Dump() {
    printf("\nCUDA - Dump (nEntries: %d)", nEntries);
    CopyToCpuIfDirty();

    std::cout << nEntries << std::endl;
    for (G4int i = 0; i < nEntries; i++) {
        printf("%0.5e %0.5e\n", h_theData[i].GetX(), h_theData[i].GetY());
    }
    std::cout << std::endl;
}

// TODO: Parallelize
void G4ParticleHPVector_CUDA::ThinOut(G4double precision) {
    if (GetVectorLength() == 0) {
        return;
    }
    CopyToCpuIfDirty();
    G4ParticleHPDataPoint * aBuff = new G4ParticleHPDataPoint[nPoints];

    G4double x, x1, x2, y, y1, y2;
    G4int count = 0, current = 2, start = 1;

    // first element always goes and is never tested
    aBuff[0] = h_theData[0];

    // find the rest
    while (current < GetVectorLength()) {
        x1 = aBuff[count].GetX();
        y1 = aBuff[count].GetY();
        x2 = h_theData[current].GetX();
        y2 = h_theData[current].GetY();
        for (G4int j = start; j < current; j++) {
            x = h_theData[j].GetX();
            if (x1 - x2 == 0) {
                y = (y2 + y1) / 2.;
            }
            else {
                y = theInt.Lin(x, x1, x2, y1, y2);
            }
            if (std::abs(y - h_theData[j].GetY()) > precision * y) {
                aBuff[++count] = h_theData[current-1]; // for this one, everything was fine
                start = current; // the next candidate
                break;
            }
        }
        current++ ;
    }

    // the last one also always goes, and is never tested
    aBuff[++count] = h_theData[GetVectorLength()-1];
    nEntries = count + 1;

    hipFree(d_theData);
    hipMalloc((void**)&d_theData, nPoints * sizeof(G4ParticleHPDataPoint));
    hipMemcpy(d_theData, aBuff, nEntries * sizeof(G4ParticleHPDataPoint), hipMemcpyHostToDevice);
    delete [] aBuff;
    isDataDirtyHost = true;
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::Merge(G4InterpolationScheme aScheme, G4double aValue,
    G4ParticleHPVector_CUDA * active, G4ParticleHPVector_CUDA * passive) {
    printf("MERGE NOT YET IMPLEMENTED\n\n");
}

__device__ int SampleGetFirstIndex_CUDA(G4double * theIntegral, G4double myRand, G4int nEntries) {
    for (int i = 0; i < nEntries; i++) {
        if (theIntegral[i] > myRand) {
            return i;
        }
    }
    return -1;
}

__global__ void SampleGetResult_CUDA(G4ParticleHPDataPoint * theData, G4double * theIntegral,
                                     G4int nEntries, G4double * result) {
    G4double myRand;
    G4double value;
    G4double test;

    G4int jcounter = 0;
    G4int jcounter_max = 1024;
    do {
        jcounter++;
        if (jcounter > jcounter_max) {
            printf("Loop-counter exceeded the threshold value.\n");
            break;
        }
        myRand = rand_CUDA();
        G4int ibin = SampleGetFirstIndex_CUDA(theIntegral, myRand, nEntries);

        if (ibin < 0) {
            printf("TKDB 080807 %f\n", myRand);
        }

        // result
        myRand = rand_CUDA();
        G4double x1, x2;
        if (ibin == 0) {
            x1 = theData[ibin].energy;
            value = x1;
            break;
        }
        else {
            x1 = theData[ibin-1].energy;
        }

        x2 = theData[ibin].energy;
        value = myRand * (x2 - x1) + x1;

        // EMendoza - Always linear interpolation:
        G4double y1 = theData[ibin-1].xSec;
        G4double y2 = theData[ibin].xSec;
        G4double mval = (y2-y1) / (x2-x1);
        G4double bval = y1 - mval * x1;
        test = (mval * value + bval) / max(theData[ibin-1].xSec, theData[ibin].xSec);
    } while (rand_CUDA() > test);
    *(result) = value;
}

G4double G4ParticleHPVector_CUDA::Sample() {
    G4double result;

    int nBlocks = GetNumBlocks(nEntries);
    SetAllNegativeXsecToZero_CUDA<<<nBlocks,THREADS_PER_BLOCK>>> (d_theData, nEntries);
    isDataDirtyHost = true;

    if (GetVectorLength() == 0) {
        return 0.0;
    }
    else if (GetVectorLength() == 1) {
        hipMemcpy(&result, &d_theData[0].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    }
    else {
        if (d_theIntegral == 0) {
            IntegrateAndNormalise();
        }
        SampleGetResult_CUDA<<<1, 1>>> (d_theData, d_theIntegral, nEntries, d_singleDoubleResult);
        hipMemcpy(&result, d_singleDoubleResult, sizeof(G4double), hipMemcpyDeviceToHost);
    }

    return result;
 }

G4double G4ParticleHPVector_CUDA::Get15percentBorder() {
    if (the15percentBorderCash > -DBL_MAX/2.) {
      return the15percentBorderCash;
    }

    CopyToCpuIfDirty();

    G4double result;
    if (GetVectorLength() == 0) {
        result = 0.0;
    }
    else if (GetVectorLength() == 1) {
      result = h_theData[0].GetX();
      the15percentBorderCash = result;
    }
    else {
      if (h_theIntegral == 0) { 
        IntegrateAndNormalise(); 
        CopyToCpuIfDirty(); // integrating changes theIntegral
      }
      G4int i;
      result = h_theData[GetVectorLength()-1].GetX();
      for (i = 0; i < GetVectorLength(); i++) {
        if (h_theIntegral[i] / h_theIntegral[GetVectorLength()-1] > 0.15) {
          result = h_theData[std::min(i+1, GetVectorLength()-1)].GetX();
          the15percentBorderCash = result;
          break;
        }
      }
      the15percentBorderCash = result;
    }
    return result;
}

G4double G4ParticleHPVector_CUDA::Get50percentBorder() {
    if (the50percentBorderCash > -DBL_MAX/2.) {
        return the50percentBorderCash;
    }
    CopyToCpuIfDirty();
    
    G4double result;
    if (GetVectorLength() == 0) {
        result = 0.0;
    } else if (GetVectorLength() == 1) {
        result = h_theData[0].GetX();
        the50percentBorderCash = result;
    } else {
        if (h_theIntegral == 0) { 
            IntegrateAndNormalise();
            CopyToCpuIfDirty(); // integrating changes theIntegral
        }
        
        G4int i;
        G4double x = 0.5;
        result = h_theData[GetVectorLength()-1].GetX();
        
        for (i = 0; i < GetVectorLength(); i++) {
            if (h_theIntegral[i] / h_theIntegral[GetVectorLength()-1] > x) {
                G4int it;
                it = i;
                if (it == GetVectorLength()-1) {
                    result = h_theData[GetVectorLength()-1].GetX();
                }
                else {
                    G4double x1, x2, y1, y2;
                    x1 = h_theIntegral[i-1] / h_theIntegral[GetVectorLength()-1];
                    x2 = h_theIntegral[i] / h_theIntegral[GetVectorLength()-1];
                    y1 = h_theData[i-1].GetX();
                    y2 = h_theData[i].GetX();
                    result = theLin.Lin(x, x1, x2, y1, y2);
                }
                the50percentBorderCash = result;
                break;
            }
        }
        the50percentBorderCash = result;
    }
    return result;
}

void G4ParticleHPVector_CUDA::Check(G4int i) {
    if (i > nEntries) {
  		// throw G4HadronicException(__FILE__, __LINE__, "Skipped some index numbers in G4ParticleHPVector");
        return;
    }
    if (i == nPoints) {
        nPoints = static_cast<G4int>(1.2 * nPoints);

        G4ParticleHPDataPoint * d_newTheData;
        hipMalloc((void**)&d_newTheData, nPoints * sizeof(G4ParticleHPDataPoint));

        int nBlocks = GetNumBlocks(nEntries);
        CopyDataPointsToBuffer_CUDA<<<nBlocks,THREADS_PER_BLOCK>>> (d_theData, d_newTheData, nEntries);

        hipFree(d_theData);
        d_theData = d_newTheData;
    }

    if (i == nEntries) {
        nEntries = i + 1;
    }

    // don't set data dirty as we haven't changed anything
}

// Geant4 doesn't ever assign private variable theBlocked,
// which means their IsBlocked function always returns false
G4bool G4ParticleHPVector_CUDA::IsBlocked(G4double aX) {
    return false;
}

G4double G4ParticleHPVector_CUDA::GetUniformRand() {
    return (G4double)rand() / (G4double)RAND_MAX;
}
