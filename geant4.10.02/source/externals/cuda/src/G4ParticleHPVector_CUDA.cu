#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"

/***********************************************
*   CUDA functions
***********************************************/
__global__ void firstIndexGreaterThan(G4ParticleHPDataPoint * theDataArg, G4double e, int* resultIndex) {
    int startIndex = blockDim.x * blockIdx.x * threadIdx.x;
    if (theDataArg[startIndex].energy > e) {
        atomicMin(resultIndex, startIndex);
    }
}

/***********************************************
*   Constructors, Deconstructors
***********************************************/
G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA()      { 
    nPoints = 20;
    hipMalloc(&theData, nPoints*sizeof(G4ParticleHPDataPoint));
    nEntries = 0;
    Verbose = 0;
    theIntegral = 0; // TODO: cuda malloc ?
    totalIntegral = -1;
    isFreed = 0;
    maxValue = -DBL_MAX;
    the15percentBorderCash = -DBL_MAX;
    the50percentBorderCash = -DBL_MAX;
    label = -DBL_MAX;
}

G4ParticleHPVector_CUDA::G4ParticleHPVector_CUDA(int n) { 
    nPoints = std::max(n,20);
    hipMalloc(&theData, nPoints*sizeof(G4ParticleHPDataPoint));
    nEntries = 0;
    Verbose = 0;
    theIntegral = 0; // TODO: cuda malloc ?
    totalIntegral = -1;
    isFreed = 0;
    maxValue = -DBL_MAX;
    the15percentBorderCash = -DBL_MAX;
    the50percentBorderCash = -DBL_MAX;
    label = -DBL_MAX;
}

G4ParticleHPVector_CUDA::~G4ParticleHPVector_CUDA() {
    if (theData) {
        hipFree(theData);
        theData = NULL;
    }
    if (theIntegral) {
       hipFree(theIntegral);
       theIntegral = NULL;
    }
    isFreed = 1;
}

/******************************************
* Getters from .hh that use CUDA
******************************************/
// TODO: check for memory leak
const G4ParticleHPDataPoint & G4ParticleHPVector_CUDA::GetPoint(G4int i) {
    G4ParticleHPDataPoint* point;
    hipMemcpy(point, &theData[i], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    return *(point);
}

G4double G4ParticleHPVector_CUDA::GetEnergy(G4int i) {
    G4double energy;
    hipMemcpy(&energy, &theData[i].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    return energy;
}

G4double G4ParticleHPVector_CUDA::GetX(G4int i) {
    G4double energy;
    hipMemcpy(&energy, &theData[i].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    return energy;
}

G4double G4ParticleHPVector_CUDA::GetXsec(G4int i) {
    G4double xSec;
    hipMemcpy(&xSec, &theData[i].xSec, sizeof(G4double), hipMemcpyDeviceToHost);
    return xSec;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::GetXsec(G4double e, G4int min) {
    return 0;
}

G4double G4ParticleHPVector_CUDA::GetY(G4double x) {
    return GetXsec(x);
}

G4double G4ParticleHPVector_CUDA::GetY(G4int i) {
    G4double xSec;
    hipMemcpy(&xSec, &theData[i].xSec, sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    return xSec;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::GetMeanX() {
    return 0;
}

/******************************************
* Setters from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::SetData(G4int i, G4double x, G4double y) {
    G4ParticleHPDataPoint point;
    point.energy = x;
    point.xSec = y;
    hipMemcpy(&theData[i], &point, sizeof(G4ParticleHPDataPoint), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetX(G4int i, G4double e) {
    hipMemcpy(&theData[i].energy, &e, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetEnergy(G4int i, G4double e) {
    hipMemcpy(&theData[i].energy, &e, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetY(G4int i, G4double x) {
    hipMemcpy(&theData[i].xSec, &x, sizeof(G4double), hipMemcpyHostToDevice);
}

void G4ParticleHPVector_CUDA::SetXsec(G4int i, G4double x) {
    hipMemcpy(&theData[i].xSec, &x, sizeof(G4double), hipMemcpyHostToDevice);
}


/******************************************
* Computations from .hh that use CUDA
******************************************/
void G4ParticleHPVector_CUDA::Init(std::istream & aDataFile, G4double ux, G4double uy) {
    G4int total;
    aDataFile >> total;
    if (theData) {
        hipFree(theData);
    }
    hipMalloc(&theData, sizeof(G4ParticleHPDataPoint) * total);
    nPoints = total;
    nEntries = 0;
    theManager.Init(aDataFile);
    Init(aDataFile, total, ux, uy);
}

void G4ParticleHPVector_CUDA::CleanUp() {
    nEntries = 0;
    theManager.CleanUp();
    maxValue = -DBL_MAX;
    if (theIntegral) {
        hipFree(theIntegral);
        theIntegral = NULL;
    }
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::SampleLin() {
    G4double result;
    if (!theIntegral) {
        IntegrateAndNormalise();
    }

    if (GetVectorLength() == 1) {
        hipMemcpy(&result, &theData[0].energy, sizeof(G4double), hipMemcpyDeviceToHost);
    }
    else {
        G4int i;
        G4double randNum = (G4double)rand() / (G4double)RAND_MAX; // TODO: change to G4UniformRand
        // TODO: requires 'first occurence' algorithm
        // for (i = GetVectorLength() - 1; i >= 0; i--) {
        //     if (randNum > )
        // }
    }

    return result;
}

// TODO: Port Me (should return theIntegral, but how do we return somethign we don't have ref to?)
G4double * G4ParticleHPVector_CUDA::Debug() {
    return 0;
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::Integrate() {
    G4int i;
    if (nEntries == 1) {
        totalIntegral = 0;
        return;
    }

    G4double sum = 0;
    // cudaIntegrate<<<1, nEntries>>>(&sum);
    totalIntegral = sum;
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::IntegrateAndNormalise() {

}

__global__ void cudaTimes(G4double factor, G4ParticleHPDataPoint* theDataArg, G4double* theIntegralArg) {
    int tid = blockIdx.x;
    theDataArg[tid].xSec = theDataArg[tid].xSec*factor;
    theIntegralArg[tid] = theIntegralArg[tid]*factor;
}
void G4ParticleHPVector_CUDA::Times(G4double factor) {
    cudaTimes<<<1, nPoints>>> (factor, theData, theIntegral);
}

/******************************************
* Functions from .cc
******************************************/
// TODO: Port Me
G4double G4ParticleHPVector_CUDA::GetXsec(G4double e) {
    int *resultIndex;
    hipMalloc(&resultIndex, sizeof(int));
    hipMemcpy(&resultIndex, &nEntries, sizeof(int), hipMemcpyHostToDevice);
    
    firstIndexGreaterThan<<<1, nEntries>>> (theData, e, resultIndex);
    
    G4int i = 0;
    hipMemcpy(&i, resultIndex, sizeof(G4int), hipMemcpyDeviceToHost);
    G4double resultVal = 0;
    hipMemcpy(&resultVal, &theData[i].xSec, sizeof(G4int), hipMemcpyDeviceToHost);
    
    G4int low = i - 1;
    G4int high = i;
    if (i == 0) {
        low = 0;
        high = 1;
    }
    else if (i == nEntries) {
        low = nEntries - 2;
        high = nEntries - 1;
    }

    G4double y;
    G4ParticleHPDataPoint pointNentriesMinusOne;
    hipMemcpy(&pointNentriesMinusOne, &theData[nEntries-1], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    
    if (e < pointNentriesMinusOne.energy) {
        G4ParticleHPDataPoint theDataLow;
        G4ParticleHPDataPoint theDataHigh;
        hipMemcpy(&theDataLow, &theData[low], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
        hipMemcpy(&theDataHigh, &theData[high], sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);

        if((theDataHigh.energy - theDataLow.energy) / theDataHigh.energy < 0.000001) {
            y = theDataLow.xSec;
        }
        else {
            y = theInt.Interpolate(theManager.GetScheme(high), e, 
                theDataLow.energy, theDataHigh.energy,
                theDataLow.xSec, theDataHigh.xSec);
        }
    }
    else {
        y = pointNentriesMinusOne.xSec;
    }

    return y;
}

void G4ParticleHPVector_CUDA::Dump() {
    G4ParticleHPDataPoint *localTheData = (G4ParticleHPDataPoint*)malloc(nPoints * sizeof(G4ParticleHPDataPoint));
    hipMemcpy(localTheData, theData, nPoints * sizeof(G4ParticleHPDataPoint), hipMemcpyDeviceToHost);
    
    std::cout << nEntries << std::endl;
    for (G4int i = 0; i < nPoints; i++) {
        std::cout << localTheData[i].GetX() << " ";
        std::cout << localTheData[i].GetY() << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;

    free(localTheData);
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::ThinOut(G4double precision) {

}

// TODO: Port Me
void G4ParticleHPVector_CUDA::Merge(G4InterpolationScheme aScheme, G4double aValue, G4ParticleHPVector_CUDA * active, G4ParticleHPVector_CUDA * passive) {

}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Sample() {
    return 0;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Get15percentBorder() {
    return 0;
}

// TODO: Port Me
G4double G4ParticleHPVector_CUDA::Get50percentBorder() {
    return 0;
}

// TODO: Port Me
void G4ParticleHPVector_CUDA::Check(G4int i) {

}

// Note: Geant4 doesn't ever assign private variable theBlocked,
// which means their IsBlocked function always returns false
G4bool G4ParticleHPVector_CUDA::IsBlocked(G4double aX) {
    return false;
}

// G4ParticleHPVector_CUDA:: & operatorPlus (G4ParticleHPVector & left, G4ParticleHPVector & right) { }
