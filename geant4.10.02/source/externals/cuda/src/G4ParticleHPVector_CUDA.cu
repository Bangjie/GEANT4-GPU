#include "hip/hip_runtime.h"
// #include <time.h>
// #include <sys/time.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include "G4ParticleHPVector_CUDA.hh"
// #include <thrust/device_vector.h>
// #include <stdio.h>
// #include <iostream>
// #include <math.h>

// __global__ void SetArrayTo(int *resArray, int numQueries, int setValue)
// {
//   int idx = blockDim.x*blockIdx.x + threadIdx.x;
//   if (idx < numQueries) {
//     resArray[idx] = setValue;
//   }
// }

// __global__ void findMinArray2(G4ParticleHPDataPoint *theData_d, G4double *queryArray_d, int *resArray_d, int numThreads, int numQueries, int nEntries)
// {
//   int idx = blockDim.x*blockIdx.x + threadIdx.x;
//   for (int i = 0; i < numQueries; i++) {
//     G4double queryEnergy = queryArray_d[i];
    
//     // search through data points in thread's range 
//     for (int j = idx; j <= nEntries; j+= numThreads) {
//       if (theData_d[j].energy >  queryEnergy) {
//         atomicMin(&resArray_d[i], j);
//         break;
//       }
//     }
  
//   }

//   // slower, 13s for highest test (and seg fault too)
//   // int start = blockIdx.x * queriesPerBlock;
//   // int i = start;
//   // do {
//   //   G4double queryEnergy = queryArray_d[i];

//   //   for (int j = idx; j < nEntries; j += numThreads) {
//   //     if (theData_d[j].energy >  queryEnergy) {
//   //       atomicMin(&resArray_d[i], j);
//   //       break;
//   //     }
//   //   }
//   //   i = ++i % numQueries;
//   // } while (i != start);
// }

// /***********************************************
// *   Device Methods
// ***********************************************/
// void G4ParticleHPVector_CUDA::SetInterpolationManager(G4InterpolationManager & aManager) {
//   theManager = aManager;
// }
// void G4ParticleHPVector_CUDA::SetInterpolationManager(const G4InterpolationManager & aManager) {
//   theManager = aManager;
// }

// double getWallTime() {
//   struct timeval time;
//   gettimeofday(&time, NULL);
//   return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
// }

// /***********************************************
// *   Host Methods
// ***********************************************/
// void G4ParticleHPVector_CUDA::GetXsecList(G4double* energiesIn_xSecsOut, G4int numQueries, G4ParticleHPDataPoint* theData, G4int nEntries) {  
//   if (nEntries == 0) {
//     for (int i = 0; i < numQueries; i++) {
//       energiesIn_xSecsOut[i] = 0.0;
//     }
//     return;
//   }

//   G4ParticleHPDataPoint * d_theData;
//   G4double              * d_energiesIn_xSecsOut;
//   G4int                 * d_minIndices;
  
//   hipMalloc((void**)&d_theData,             sizeof(G4ParticleHPDataPoint)            * nEntries);
//   hipMalloc((void**)&d_energiesIn_xSecsOut, sizeof(G4double)            * numQueries);
//   hipMalloc((void**)&d_minIndices,          sizeof(G4int)               * numQueries);
//   G4int *minIndices = (G4int*)malloc(numQueries * sizeof(G4int));

//   hipMemcpy(d_theData, theData, sizeof(G4ParticleHPDataPoint) * nEntries, hipMemcpyHostToDevice);
//   hipMemcpy(d_energiesIn_xSecsOut, energiesIn_xSecsOut, sizeof(G4double) * numQueries, hipMemcpyHostToDevice);
  
//   int queryBlocks = numQueries/THREADS_PER_BLOCK + (numQueries % THREADS_PER_BLOCK == 0 ? 0:1);
//   int dataChunk = 1;
//   int threadNum = nEntries/dataChunk;
//   int arrayBlocks = threadNum/THREADS_PER_BLOCK + (threadNum % THREADS_PER_BLOCK == 0 ? 0:1);
//   int queriesPerBlock = numQueries / arrayBlocks;
  
//   double a = getWallTime();
//   SetArrayTo <<< queryBlocks, THREADS_PER_BLOCK >>>(d_minIndices, numQueries, nEntries-1);
//   findMinArray2 <<< arrayBlocks, THREADS_PER_BLOCK >>> (d_theData, d_energiesIn_xSecsOut, d_minIndices, threadNum, numQueries, nEntries);
//   hipDeviceSynchronize();
//   printf("Time (nEntries = %d, numQueries = %d): %f\n", nEntries, numQueries, getWallTime() - a);
  
//   hipMemcpy(minIndices, d_minIndices, numQueries * sizeof(G4int), hipMemcpyDeviceToHost);

//   for (int i = 0; i < numQueries; i++) {
//     int minIndex = minIndices[i];
   
//     G4int low = minIndex - 1;
//     G4int high = minIndex;
//     G4double e = energiesIn_xSecsOut[i];
    
//     if (minIndex == 0)
//     {
//       low = 0;
//       high = 1;
//     }
//     else if (minIndex == nEntries)
//     {
//       low = nEntries - 2;
//       high = nEntries - 1;
//     }

//     if (e < theData[nEntries-1].GetX())
//     {
//       if (theData[high].GetX() != 0 
//         && (std::abs((theData[high].GetX() - theData[low].GetX()) / theData[high].GetX()) < 0.000001))
//       {
//         energiesIn_xSecsOut[i] = theData[low].GetY();
//       }
//       else
//       {
//         energiesIn_xSecsOut[i] = 
//           theInt.Interpolate(theManager.GetScheme(high), e, 
//                              theData[low].GetX(), theData[high].GetX(),
//                              theData[low].GetY(), theData[high].GetY());
//       }
//     }
//     else
//     {
//       energiesIn_xSecsOut[i] = theData[nEntries-1].GetY();
//     }
//   }

//   hipFree(d_theData);
//   hipFree(d_energiesIn_xSecsOut);
//   hipFree(d_minIndices);
//   free(minIndices);
// }


#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "G4ParticleHPVector_CUDA.hh"

/***********************************************
*   Device Methods
***********************************************/
__global__ void GetMinIndices_CUDA(G4ParticleHPDataPoint *d_theData, int nEntries, 
                                   double *d_energiesIn_xSecsOut, int numQueries, int *d_minIndices) {
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int stepSize = (int)sqrt((float)nEntries);

  if (idx < numQueries) {
    int i = 0;
    double e = d_energiesIn_xSecsOut[idx];
    
    for (i = 0; i < nEntries; i += stepSize) {
      if (d_theData[i].energy >= e) {
        break;
      }
    }
    
    i = (i - (stepSize - 1) >= 0) ? i - (stepSize - 1) : 0; 
    for (; i < nEntries; i++) {
      if (d_theData[i].energy >= e) {
        break;
      }
    }

    d_minIndices[idx] = i;
  }
}

void G4ParticleHPVector_CUDA::SetInterpolationManager(G4InterpolationManager & aManager) {
  theManager = aManager;
}
void G4ParticleHPVector_CUDA::SetInterpolationManager(const G4InterpolationManager & aManager) {
  theManager = aManager;
}

double getWallTime() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
}

/***********************************************
*   Host Methods
***********************************************/
void G4ParticleHPVector_CUDA::GetXsecList(G4double* energiesIn_xSecsOut, G4int numQueries, G4ParticleHPDataPoint* theData, G4int nEntries) {  
  if (nEntries == 0) {
    for (int i = 0; i < numQueries; i++) {
      energiesIn_xSecsOut[i] = 0.0;
    }
    return;
  }

  G4ParticleHPDataPoint * d_theData;
  G4double              * d_energiesIn_xSecsOut;
  G4int                 * d_minIndices;
  
  hipMalloc((void**)&d_theData,             sizeof(G4ParticleHPDataPoint)            * nEntries);
  hipMalloc((void**)&d_energiesIn_xSecsOut, sizeof(G4double)            * numQueries);
  hipMalloc((void**)&d_minIndices,          sizeof(G4int)               * numQueries);
  G4int *minIndices = (G4int*)malloc(numQueries * sizeof(G4int));

  hipMemcpy(d_theData, theData, sizeof(G4ParticleHPDataPoint) * nEntries, hipMemcpyHostToDevice);
  hipMemcpy(d_energiesIn_xSecsOut, energiesIn_xSecsOut, sizeof(G4double) * numQueries, hipMemcpyHostToDevice);
  
  // need to add 1 block if doesn't divide evenly (e.g 32 T_P_B, 36 numQueries we need 1+1=2 blocks to get those last 4 queries)
  int numBlocksSingleElement = numQueries/THREADS_PER_BLOCK + ((numQueries % THREADS_PER_BLOCK == 0) ? 0 : 1);

  GetMinIndices_CUDA <<<numBlocksSingleElement, THREADS_PER_BLOCK>>>
    (d_theData, nEntries, d_energiesIn_xSecsOut, numQueries, d_minIndices);

  hipMemcpy(minIndices, d_minIndices, sizeof(G4int) * numQueries, hipMemcpyDeviceToHost);

  for (int i = 0; i < numQueries; i++) {
    int minIndex = minIndices[i];
   
    G4int low = minIndex - 1;
    G4int high = minIndex;
    G4double e = energiesIn_xSecsOut[i];
    
    if (minIndex == 0)
    {
      low = 0;
      high = 1;
    }
    else if (minIndex == nEntries)
    {
      low = nEntries - 2;
      high = nEntries - 1;
    }

    if (e < theData[nEntries-1].GetX())
    {
      if (theData[high].GetX() != 0 
        &&(std::abs((theData[high].GetX() - theData[low].GetX()) / theData[high].GetX()) < 0.000001))
      {
        energiesIn_xSecsOut[i] = theData[low].GetY();
      }
      else
      {
        energiesIn_xSecsOut[i] = theInt.Interpolate(theManager.GetScheme(high), e, 
                               theData[low].GetX(), theData[high].GetX(),
                               theData[low].GetY(), theData[high].GetY());
      }
    }
    else
    {
      energiesIn_xSecsOut[i] = theData[nEntries-1].GetY();
    }
  }

  hipFree(d_theData);
  hipFree(d_energiesIn_xSecsOut);
  hipFree(d_minIndices);
  free(minIndices);
}