#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "CUDA_G4NeutronHPVector.h"

// CUDA kernel
__global__
void sumArrays(int* arr1, int* arr2, int* res, int n)
{
  int tid = blockIdx.x;
  if (tid < n) 
      res[tid] = arr1[tid] + arr2[tid];
}

void CUDA_sumArrays(int* arr1, int* arr2, int* res, int n) {
    int *gpu_arr1, *gpu_arr2, *gpu_res;

    hipMalloc((void**)&gpu_arr1, n*sizeof(int));
    hipMalloc((void**)&gpu_arr2, n*sizeof(int));
    hipMalloc((void**)&gpu_res, n*sizeof(int));
    
    hipMemcpy(gpu_arr1, arr1, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_arr2, arr2, n*sizeof(int), hipMemcpyHostToDevice);

    sumArrays<<<n,1>>>(gpu_arr1, gpu_arr2, gpu_res, n);

    hipMemcpy(res, gpu_res, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpu_arr1);
    hipFree(gpu_arr2);
    hipFree(gpu_res);
}
