#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "DeviceMain.h"
#include <vector>

class ToyClass
{
	public:
	int* data;
	
	ToyClass(int x)
	{
		data = new int[1];
		data[0] = x;
	}
	void add_one()
	{
		data[0] = data[0] + 1;
	}
};

__global__ void useClass(ToyClass *toyClass)
{
	printf("%d\n", toyClass->data[0]);
}

int main()
{
	ToyClass c(1);
	// create class storage on device and copy top level class
	ToyClass *d_c;
	hipMalloc((void **)&d_c, sizeof(ToyClass));
	hipMemcpy(d_c, &c, sizeof(ToyClass), hipMemcpyHostToDevice);
	// make an allocated region on device for use by pointer in class
	int *hostdata;
	hipMalloc((void **)&hostdata, sizeof(int));
	hipMemcpy(hostdata, c.data, sizeof(int), hipMemcpyHostToDevice);
	// copy pointer to allocated device storage to device class
	hipMemcpy(&(d_c->data), &hostdata, sizeof(int *), hipMemcpyHostToDevice);
	useClass<<<1,1>>>(d_c);
	hipDeviceSynchronize();
	
	return 0;



}